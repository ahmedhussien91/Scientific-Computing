#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <fstream>
#include <chrono>

using namespace std;
using namespace std::chrono;

#define  MAX_NUM_OF_ROWS 1024
#define	 MAX_NUM_OF_COLS 1024

float Matrix_1[MAX_NUM_OF_ROWS][MAX_NUM_OF_COLS];
float Matrix_2[MAX_NUM_OF_ROWS][MAX_NUM_OF_COLS];
float Matrix_output[MAX_NUM_OF_ROWS][MAX_NUM_OF_COLS];

int no_of_rows_1 = 4;
int no_of_rows_2 = 4;
int	no_of_cols_1 = 4;
int	no_of_cols_2 = 4;

void ReadMatrix_1_2(void);
hipError_t MultiplyWithCuda(void);

__global__ void MultiplyKernel(float *c, const float *a, const float *b, const int wc, const int hc, const int CommonDim)
{
	// Block index
	int bx = blockIdx.x;
	int by = blockIdx.y;

	// Thread index
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	const int BLOCK_SIZE = 32;

	// Index of the first sub-matrix of A processed by the block
	int aBegin = wc * BLOCK_SIZE * by;

	// Index of the last sub-matrix of A processed by the block
	int aEnd = aBegin + wc - 1;

	// Step size used to iterate through the sub-matrices of A
	int aStep = BLOCK_SIZE;

	// Index of the first sub-matrix of B processed by the block
	int bBegin = BLOCK_SIZE * bx;

	// Step size used to iterate through the sub-matrices of B
	int bStep = BLOCK_SIZE * CommonDim;

	// Csub is used to store the element of the block sub-matrix
	// that is computed by the thread
	float Csub = 0;

	// Loop over all the sub-matrices of A and B
	// required to compute the block sub-matrix
	for (int Sa = aBegin, Sb = bBegin;
		Sa <= aEnd;
		Sa += aStep, Sb += bStep) {
		// Declaration of the shared memory array As used to
		// store the sub-matrix of A
		__shared__ float As[BLOCK_SIZE][BLOCK_SIZE];

		// Declaration of the shared memory array Bs used to
		// store the sub-matrix of B
		__shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

		// Load the matrices from device memory
		// to shared memory; each thread loads
		// one element of each matrix
		As[ty][tx] = a[Sa + wc * ty + tx];
		Bs[ty][tx] = b[Sb + CommonDim * ty + tx];

		// Synchronize to make sure the matrices are loaded
		__syncthreads();

		// Multiply the two matrices together;
		// each thread computes one element
		// of the block sub-matrix
#pragma unroll

		for (int k = 0; k < BLOCK_SIZE; ++k) {
			Csub += As[ty][k] * Bs[k][tx];
		}

		// Synchronize to make sure that the preceding
		// computation is done before loading two new
		// sub-matrices of A and B in the next iteration
		__syncthreads();
	}

	// Write the block sub-matrix to device memory;
	// each thread writes one element
	int Sc = CommonDim * BLOCK_SIZE * by + BLOCK_SIZE * bx;
	c[Sc + CommonDim * ty + tx] = Csub;
}


int main()
{

	// read 2 Matrix from Files
	ReadMatrix_1_2();
	// take time snap before multiplication
	high_resolution_clock::time_point t1 = high_resolution_clock::now();
	//CPU Multiplication MAtrix_1 * Matrix_2
	hipError_t cudaStatus = MultiplyWithCuda();
	// take time snap after multiplication
	high_resolution_clock::time_point t2 = high_resolution_clock::now();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}

	// print the Time taken to Multiply two Matrices 
	auto duration = duration_cast<microseconds>(t2 - t1).count();
	cout << "Multiplication Time CPU(us):" << duration << "\n";

	//print the output matrix for testing
//	for (int i = 0; i < no_of_cols_1; i++) {
//		for (int j = 0; j < no_of_rows_2; j++)
//		{
//			cout << Matrix_output[i][j] << "	";
//		}
//		cout << "\n";
//	}

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	return 0;
}

// this function read the two matrices from two files with the dimensions given in the global variables set at the begining of the file
void ReadMatrix_1_2(void) {
	int x, y;
	ifstream in_1("Matrix_1.txt", std::ifstream::in);
	ifstream in_2("Matrix_2.txt", std::ifstream::in);

	if (!in_1 || !in_2) {
		cout << "Error! Cannot open file.\n";
		return;
	}
	else if (no_of_cols_1 != no_of_rows_2) {
		cout << "Error! Matrix dimensions is not valid for multiplication.\n";
		return;
	}

	for (y = 0; y < no_of_cols_1; y++) {
		for (x = 0; x < no_of_rows_1; x++) {
			in_1 >> Matrix_1[x][y];
		}
	}
	for (y = 0; y < no_of_cols_2; y++) {
		for (x = 0; x < no_of_rows_2; x++) {
			in_2 >> Matrix_2[x][y];
		}
	}

	in_1.close();
	in_2.close();
}


// Helper function for using CUDA to add vectors in parallel.
hipError_t MultiplyWithCuda()
{
	float *dev_a = 0; //Matrix_1
	float *dev_b = 0; //Matrix_2
	float *dev_c = 0; //Matrix_output
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_c, no_of_rows_1*no_of_cols_2 * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_a, no_of_rows_1*no_of_cols_1 * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_b, no_of_rows_2*no_of_cols_2 * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_a, Matrix_1, no_of_rows_1*no_of_cols_1 * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_b, Matrix_2, no_of_rows_2*no_of_cols_2 * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	dim3 threasPerBlock(32, 32);
	dim3 blocksPerGrid(ceil(double(no_of_rows_1) / threasPerBlock.x), ceil(double(no_of_cols_2) / threasPerBlock.y));


	// Launch a kernel on the GPU with one thread for each element.
	MultiplyKernel <<< blocksPerGrid, threasPerBlock >>> (dev_c, dev_a, dev_b, no_of_rows_1, no_of_cols_2, no_of_cols_1);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	int output_width = no_of_rows_1 * no_of_cols_2;
	cudaStatus = hipMemcpy(Matrix_output, dev_c, output_width * sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);

	return cudaStatus;
}
